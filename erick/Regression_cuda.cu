#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "FloatMatrix.h"
#include "Regression_cuda.h"
#include<bits/stdc++.h>


#define THREAD_SIZE 512
#define FSIZE sizeof(float)

using namespace std;

/*device functions */
__global__ void d_transpose(float *d_arr_tr,float *d_arr, int rs, int cs);

/*multiplies matrix srt and dest to dest */

__global__ void d_mat_mult(float *dest, float *arr_t, float *arr, int rs, int cs);
__global__ void d_swap_row(float* row1,float* row2,int rs);
__global__ void d_copy_row(float* dst, float* src, int rs);
__global__ void d_det_mult(float *mat, float *temp, float num1, float num2,int rs);


/*first transposes a matrix and then multiplies it with itself*/
FloatMatrix* transpose_mult(FloatMatrix *fm){ 
        float *d_arr_tr, *d_arr;

        int rs = fm->row_size;
        int cs = fm->col_size;
        hipMalloc((void**) &d_arr_tr, FSIZE*rs*cs);
        hipMalloc((void**) &d_arr, FSIZE*rs*cs);

        hipMemcpy(d_arr,fm->mat,FSIZE*rs*cs,hipMemcpyHostToDevice);

        int blocks = (rs*cs/THREAD_SIZE) + (rs*cs%THREAD_SIZE != 0);
        d_transpose<<<blocks,THREAD_SIZE>>>(d_arr_tr,d_arr,rs,cs);

        FloatMatrix *mult_result = new FloatMatrix(cs,cs);
        int  mult_size = cs*cs;
        float *d_mult_arr;
        hipMalloc((void**) &d_mult_arr,FSIZE*mult_size);

        blocks = mult_size/THREAD_SIZE + (rs*cs%THREAD_SIZE != 0);

        d_mat_mult<<<blocks,THREAD_SIZE>>>(d_mult_arr,d_arr_tr,d_arr,rs,cs);
        hipMemcpy(mult_result->mat,d_mult_arr,FSIZE*mult_size,hipMemcpyDeviceToHost);

        hipFree(d_arr_tr);
        hipFree(d_arr);
	hipFree(d_mult_arr);
        return mult_result; 

}

/*modified code from geeks for geeks */
float determinant_of_matrix(float *mat, int n){
	float* mat_cpy = new float[n*n];
	memcpy(mat_cpy,mat,FSIZE*n*n);
	float num1,num2, det = 1,total = 1; 
	int index;
	float *d_arr;// array for swapping rows
	float *d_temp;
	hipMalloc((void**) &d_arr, FSIZE*n*n);
	hipMalloc((void**) &d_temp,FSIZE*n +1);	
	int blocks ;
	
	float temp[n+1]; 

	for(int i = 0 ; i < n ; i++){
		index = i; 
		while(mat_cpy[index*n+i] == 0 && index < n)
			index++;
		if(index == n)
			continue;
		if(index != i){
			blocks = n/THREAD_SIZE + (n%THREAD_SIZE != 0);
			hipMemcpy(d_arr,mat_cpy,FSIZE*n*n,hipMemcpyHostToDevice);
			d_swap_row<<<blocks,THREAD_SIZE>>>(&d_arr[index*n],&d_arr[i*n],n);
			hipMemcpy(mat_cpy,d_arr,FSIZE*n*n,hipMemcpyDeviceToHost);
			det = det*pow(-1,index-i);	
		}
		
		

		memcpy(temp,&mat_cpy[i*n],FSIZE*n);
		hipMemcpy(d_temp,temp,FSIZE*n+1,hipMemcpyHostToDevice);
					
		for(int j = i+1; j < n; j++){
			
			num1 = temp[i];
			num2 = mat_cpy[j*n + i]; 
			blocks = n/THREAD_SIZE + (n%THREAD_SIZE != 0);
			hipMemcpy(&d_arr[j*n],&mat_cpy[j*n],FSIZE*n,hipMemcpyHostToDevice);
			d_det_mult<<<blocks,THREAD_SIZE>>>(&d_arr[j*n],d_temp,num1,num2,n);
			hipMemcpy(&mat_cpy[j*n],&d_arr[j*n],FSIZE*n,hipMemcpyDeviceToHost);
			
			total = total * num1;
			
		}
	}

	for(int i = 0 ; i < n; i++)
		det = det*mat_cpy[i*n +i];

	hipFree(d_arr);
	hipFree(d_temp);
	free(mat_cpy);
	return det/total;

}



__global__ void d_transpose(float *d_arr_tr,float *d_arr, int rs, int cs){

        int i = blockIdx.x*blockDim.x + threadIdx.x;

	if( i < rs*cs){
            int row = (i%cs);
            int col = (i/cs);

             int trans_index = row*(rs) + col;

             d_arr_tr[trans_index] = d_arr[i];
        }

}


__global__ void d_mat_mult(float *dest, float *arr_t, float *arr, int rs, int cs){


    int i = blockIdx.x*blockDim.x + threadIdx.x;


    int cs_t = rs;
    int rs_t = cs;

    __shared__ float shared[THREAD_SIZE];

    int d_row; 
    int d_col;
    if( i < rs_t*cs){
            shared[i] = 0;
            d_row = i/cs;
            d_col = i%cs;
    

        for(int k = 0 ; k < rs ; k++){
           shared[i] += arr_t[(d_row*cs_t)+k]*arr[(k*cs) + d_col];
            __syncthreads();
        }

        dest[i] = shared[i];


    }
 
}

__global__ void d_swap_row(float *row1,float *row2,int rs){

	__shared__ float shared[THREAD_SIZE];

    	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if( i < rs)
		shared[i] = row1[i];

	if( i < rs){
		row1[i] = row2[i];
		row2[i] = shared[i];
	}

}

__global__ void d_copy_row(float* dst, float* src, int rs){

	
    	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if( i < rs)
		dst[i] = src[i];

}

__global__ void d_det_mult(float *mat, float *temp, float num1, float num2,int rs){
	
    	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if( i < rs)
		mat[i] = (num1*mat[i]) - (num2*temp[i]);

}



