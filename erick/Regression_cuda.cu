#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "FloatMatrix.h"
#include "Regression_cuda.h"
#include<bits/stdc++.h>


#define THREAD_SIZE 512
#define FSIZE sizeof(float)

using namespace std;

/*device functions */
__global__ void d_transpose(float *d_arr_tr,float *d_arr, int rs, int cs);

/*multiplies matrix srt and dest to dest */

__global__ void d_mat_mult_t(float *dest, float *arr_t, float *arr, int rs, int cs);
__global__ void d_swap_row(float* row1,float* row2,int rs);
__global__ void d_copy_row(float* dst, float* src, int rs);
__global__ void d_det_mult(float *mat, float *temp, float num1, float num2,int rs);
__global__ void d_mat_mult(float *dest, float *x_arr, float* y_arr, int x_rs, int x_cs, int y_rs, int y_cs);


/*first transposes a matrix and then multiplies it with itself*/
FloatMatrix* transpose_x_mult_x(FloatMatrix *fm){ 
        float *d_arr_tr, *d_arr;

        int rs = fm->row_size;
        int cs = fm->col_size;
        hipMalloc((void**) &d_arr_tr, FSIZE*rs*cs);
        hipMalloc((void**) &d_arr, FSIZE*rs*cs);

        hipMemcpy(d_arr,fm->mat,FSIZE*rs*cs,hipMemcpyHostToDevice);

        int blocks = (rs*cs/THREAD_SIZE) + (rs*cs%THREAD_SIZE != 0);
        d_transpose<<<blocks,THREAD_SIZE>>>(d_arr_tr,d_arr,rs,cs);

        FloatMatrix *mult_result = new FloatMatrix(cs,cs);
        int  mult_size = cs*cs;
        float *d_mult_arr;
        hipMalloc((void**) &d_mult_arr,FSIZE*mult_size);

        blocks = mult_size/THREAD_SIZE + (mult_size%THREAD_SIZE != 0);

        d_mat_mult_t<<<blocks,THREAD_SIZE>>>(d_mult_arr,d_arr_tr,d_arr,rs,cs);
        hipMemcpy(mult_result->mat,d_mult_arr,FSIZE*mult_size,hipMemcpyDeviceToHost);

        hipFree(d_arr_tr);
        hipFree(d_arr);
	hipFree(d_mult_arr);
        return mult_result; 

}

/* Transpose a matrix and then multiply it with Y */
FloatMatrix* transpose_x_mult_y(FloatMatrix *x, FloatMatrix *y){
        float *d_arr_tr, *d_arr_x, *d_arr_y;
	int x_rs = x->row_size;
	int x_cs = x->col_size;
	int y_rs = y->row_size;
	int y_cs = y->col_size;
	int xt_rs = x_cs;
	int xt_cs = x_rs;
	
	/* cuda memory for matrix/array */
        hipMalloc((void**) &d_arr_x, FSIZE*x_rs*x_cs);
	hipMalloc((void**) &d_arr_y, FSIZE*y_rs*y_cs);
        hipMalloc((void**) &d_arr_tr, FSIZE*xt_rs*xt_cs);

	int blocks = (x_rs*x_cs/THREAD_SIZE) + ((x_rs*x_cs)%THREAD_SIZE != 0);


	/*Transposing matrix x */	
	hipMemcpy(d_arr_x,x->mat,FSIZE*x_rs*x_cs,hipMemcpyHostToDevice);
	d_transpose<<<blocks,THREAD_SIZE>>>(d_arr_tr,d_arr_x,x_rs,x_cs);	

	/*For debugging */
	/*	
	FloatMatrix *trans = new FloatMatrix(xt_rs,xt_cs);
	hipMemcpy(trans->mat,d_arr_tr,FSIZE*xt_rs*xt_cs,hipMemcpyDeviceToHost);
	trans->print();
	printf("\n");
	delete(trans);
	*/
	/*               */

	
	FloatMatrix *mult_result = new FloatMatrix(xt_rs,y_cs);	
	int mult_size = xt_rs*y_cs;
	float *d_mult_arr;
	hipMalloc((void**) &d_mult_arr,FSIZE*mult_size); 

	
	/* copies the matrix y into the cuda memory */	
	hipMemcpy(d_arr_y,y->mat,FSIZE*y_rs*y_cs,hipMemcpyHostToDevice);

	blocks = mult_size/THREAD_SIZE + (mult_size%THREAD_SIZE != 0);
	d_mat_mult<<<blocks,THREAD_SIZE>>>(d_mult_arr,d_arr_tr,d_arr_y,xt_rs,xt_cs,y_rs,y_cs);
        hipMemcpy(mult_result->mat,d_mult_arr,FSIZE*mult_size,hipMemcpyDeviceToHost);

	/*Free allocated cuda memory*/
        hipFree(d_arr_tr);
        hipFree(d_arr_x);
	hipFree(d_arr_y);
	hipFree(d_mult_arr);
	return mult_result;
}


/*modified code from geeks for geeks */
float determinant_of_matrix(float *mat, int n){
	float* mat_cpy = new float[n*n];
	memcpy(mat_cpy,mat,FSIZE*n*n);
	float num1,num2, det = 1,total = 1; 
	int index;
	float *d_arr;// array for swapping rows
	float *d_temp;
	hipMalloc((void**) &d_arr, FSIZE*n*n);
	hipMalloc((void**) &d_temp,FSIZE*n +1);	
	int blocks ;
	
	float temp[n+1]; 

	for(int i = 0 ; i < n ; i++){
		index = i; 
		while(mat_cpy[index*n+i] == 0 && index < n)
			index++;
		if(index == n)
			continue;
		if(index != i){
			blocks = n/THREAD_SIZE + (n%THREAD_SIZE != 0);
			hipMemcpy(d_arr,mat_cpy,FSIZE*n*n,hipMemcpyHostToDevice);
			d_swap_row<<<blocks,THREAD_SIZE>>>(&d_arr[index*n],&d_arr[i*n],n);
			hipMemcpy(mat_cpy,d_arr,FSIZE*n*n,hipMemcpyDeviceToHost);
			det = det*pow(-1,index-i);	
		}
		
		

		memcpy(temp,&mat_cpy[i*n],FSIZE*n);
		hipMemcpy(d_temp,temp,FSIZE*n+1,hipMemcpyHostToDevice);
					
		for(int j = i+1; j < n; j++){
			
			num1 = temp[i];
			num2 = mat_cpy[j*n + i]; 
			blocks = n/THREAD_SIZE + (n%THREAD_SIZE != 0);
			hipMemcpy(&d_arr[j*n],&mat_cpy[j*n],FSIZE*n,hipMemcpyHostToDevice);
			d_det_mult<<<blocks,THREAD_SIZE>>>(&d_arr[j*n],d_temp,num1,num2,n);
			hipMemcpy(&mat_cpy[j*n],&d_arr[j*n],FSIZE*n,hipMemcpyDeviceToHost);
			
			total = total * num1;
			
		}
	}

	for(int i = 0 ; i < n; i++)
		det = det*mat_cpy[i*n +i];

	hipFree(d_arr);
	hipFree(d_temp);
	free(mat_cpy);
	return det/total;

}


/* Modified code from geeks for geeks, URL: https://www.geeksforgeeks.org/adjoint-inverse-matrix/ */
void getCofactor(float *mat, float *temp, int p, int q, int n){ 
    int i = 0, j = 0; 
  
    // Looping for each element of the matrix 
    for (int row = 0; row < n; row++) 
    { 
        for (int col = 0; col < n; col++) 
        { 
            //  Copying into temporary matrix only those element 
            //  which are not in given row and column 
            if (row != p && col != q) 
            { 
                temp[i*(n-1)+j] = mat[row*n+col]; 
		j++;
  
                // Row is filled, so increase row index and 
                // reset col index 
                if (j == n - 1) 
                { 
                    j = 0; 
                    i++; 
                } 
            } 
        } 
    } 
} 


/* Modified code from geeks for geeks, URL : https://www.geeksforgeeks.org/adjoint-inverse-matrix/ */
float* adjoint(float* mat, int n){ 
    float *adj = new float[n*n];
    if (n == 1) 
    { 
        adj[0] = 1; 
        return adj; 
    } 
  
    // temp is used to store cofactors of A[][] 
    int sign = 1;
    float *temp = new float[n*n];
  
    for (int i=0; i<n; i++) 
    { 
        for (int j=0; j<n; j++) 
        { 
            // Get cofactor of A[i][j] 
            getCofactor(mat, temp, i, j, n); 
  
            // sign of adj[j][i] positive if sum of row 
            // and column indexes is even. 
            sign = ((i+j)%2==0)? 1: -1; 
  
            // Interchanging rows and columns to get the 
            // transpose of the cofactor matrix 
            adj[j*n+i] = (sign)*(determinant_of_matrix(temp, n-1)); 
        } 
    } 
    free(temp);
    return adj;
} 

/* code modified from geeks for geeks URL: // Function to calculate and store inverse, returns false if */
// matrix is singular 
float* inverse_of_matrix(float* mat, int n) 
{ 

    float *inv = new float[n*n];
    float det = determinant_of_matrix(mat, n); 
    if (det == 0) 
    { 
        cout << "Singular matrix, can't find its inverse"; 
        return NULL; 
    } 
  
    // Find adjoint 
    float *adj = adjoint(mat, n); 
  
    // Find Inverse using formula "inverse(A) = adj(A)/det(A)" 
    for (int i=0; i<n; i++) 
        for (int j=0; j<n; j++) 
            inv[i*n +j] = adj[i*n+j]/det; 
    free(adj);
    return inv;
  
} 

__global__ void d_transpose(float *d_arr_tr,float *d_arr, int rs, int cs){

        int i = blockIdx.x*blockDim.x + threadIdx.x;

	if( i < rs*cs){
            int row = (i%cs);
            int col = (i/cs);

             int trans_index = row*(rs) + col;

             d_arr_tr[trans_index] = d_arr[i];
        }

}


__global__ void d_mat_mult_t(float *dest, float *arr_t, float *arr, int rs, int cs){


    int i = blockIdx.x*blockDim.x + threadIdx.x;


    int cs_t = rs;
    int rs_t = cs;

    __shared__ float shared[THREAD_SIZE];

    int d_row; 
    int d_col;
    if( i < rs_t*cs){
            shared[i] = 0;
            d_row = i/cs;
            d_col = i%cs;
    

        for(int k = 0 ; k < rs ; k++){
           shared[i] += arr_t[(d_row*cs_t)+k]*arr[(k*cs) + d_col];
            __syncthreads();
        }

        dest[i] = shared[i];


    }
 
}

__global__ void d_mat_mult(float *dest, float *x_arr, float* y_arr, int x_rs, int x_cs, int y_rs, int y_cs){

	int  i = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ float shared[THREAD_SIZE];
	int d_row,d_col;
	if( i < x_rs*y_cs){
		shared[i] = 0;
		d_row = i/x_cs;
		d_col = i%y_cs;
	
	
		for(int k = 0 ; k < y_rs ;k++){
			shared[i] += x_arr[(d_row*x_cs)+k] * y_arr[(k*y_cs) + d_col];
			__syncthreads();
		}
		
		dest[i] = shared[i];

	}
	
}

__global__ void d_swap_row(float *row1,float *row2,int rs){

	__shared__ float shared[THREAD_SIZE];

    	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if( i < rs)
		shared[i] = row1[i];

	if( i < rs){
		row1[i] = row2[i];
		row2[i] = shared[i];
	}

}

__global__ void d_copy_row(float* dst, float* src, int rs){

	
    	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if( i < rs)
		dst[i] = src[i];

}

__global__ void d_det_mult(float *mat, float *temp, float num1, float num2,int rs){
	
    	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if( i < rs)
		mat[i] = (num1*mat[i]) - (num2*temp[i]);

}



